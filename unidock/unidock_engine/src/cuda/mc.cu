#include "hip/hip_runtime.h"
//
// Created by Congcong Liu on 24-9-25.
//

#include <hip/hip_cooperative_groups.h>

#include "myutils/myio.h"
#include "model/model.h"
#include "search/mc.h"

#include "common.cuh"
#include "myutils/errors.h"
#include "myutils/mymath.h"
#include "geometry/quaternion.h"
#include "bfgs.cuh"



__device__ __forceinline__ void randomize_pose_warp(const cg::thread_block_tile<TILE_SIZE>& tile,
                                                    FlexPose* out_pose_new, FlexPoseGradient* aux_g,
                                                    const FlexPose* pose_old, const FlexTopo& flex_topo,
                                                    int n,
                                                    hiprandStatePhilox4_32_10_t* state){
    float4 rf4 = hiprand_uniform4(state);
    Real tmp4[4] = {0};
    Real rotvec[3] = {map_01_to_dot5(rf4.x), map_01_to_dot5(rf4.y), map_01_to_dot5(rf4.z)};
    uint4 ri4 = curand4(state);

    // copy cartesian coordinates
    for (int i = tile.thread_rank(); i < flex_topo.natom * 3; i += tile.num_threads()){
        out_pose_new->coords[i] = pose_old->coords[i];
    }
    tile.sync();

    // generate a random pose inside the box
    if (tile.thread_rank() == 0){
        // set energy
        out_pose_new->energy = pose_old->energy;

        // copy center & orientation
        out_pose_new->center[0] = pose_old->center[0];
        out_pose_new->center[1] = pose_old->center[1];
        out_pose_new->center[2] = pose_old->center[2];
        out_pose_new->rot_vec[0] = pose_old->rot_vec[0];
        out_pose_new->rot_vec[1] = pose_old->rot_vec[1];
        out_pose_new->rot_vec[2] = pose_old->rot_vec[2];

        if (FLAG_CONSTRAINT_DOCK){
            // center and orientation are fixed
            aux_g->center_g[0] = 0;
            aux_g->center_g[1] = 0;
            aux_g->center_g[2] = 0;
            aux_g->orientation_g[0] = 0;
            aux_g->orientation_g[1] = 0;
            aux_g->orientation_g[2] = 0;
        }
        else{
            // random center, set gradient
            Real a = gyration_radius(pose_old, &flex_topo);
            tmp4[0] = get_real_within_by_int(ri4.x, BOX_X_LO + a, BOX_X_HI - a, ceil((BOX_X_HI - BOX_X_LO - 2 * a) / BOX_PREC) + 1);
            tmp4[1] = get_real_within_by_int(ri4.y, BOX_Y_LO + a, BOX_Y_HI - a, ceil((BOX_Y_HI - BOX_Y_LO - 2 * a) / BOX_PREC) + 1);
            tmp4[2] = get_real_within_by_int(ri4.z, BOX_Z_LO + a, BOX_Z_HI - a, ceil((BOX_Z_HI - BOX_Z_LO - 2 * a) / BOX_PREC) + 1);

            aux_g->center_g[0] = tmp4[0] - out_pose_new->center[0];
            aux_g->center_g[1] = tmp4[1] - out_pose_new->center[1];
            aux_g->center_g[2] = tmp4[2] - out_pose_new->center[2];

            // random orientation, set gradient.
            // Alexa, M. (2022). Super-Fibonacci Spirals: Fast, Low-Discrepancy Sampling of SO(3). Proceedings of the
            // IEEE Computer Society Conference on Computer Vision and Pattern Recognition, 2022-June(3), 8281–8290.
            // https://doi.org/10.1109/CVPR52688.2022.00811

            int id_pose = ((blockIdx.x * blockDim.x + threadIdx.x) % (n * TILE_SIZE)) / TILE_SIZE; // global idx of the thread
            DPrint1("id_pose: %d\n", id_pose);

            Real s = id_pose + 0.5;
            Real r = sqrt( s / n);
            Real R = sqrt(1.0 - s / n);
            Real alpha = 2.0 * PI * s / PHI;
            Real beta = 2.0 * PI * s / PSI;
            tmp4[0] = r * sin(alpha);
            tmp4[1] = r * cos(alpha);
            tmp4[2] = R * sin(beta);
            tmp4[3] = R * cos(beta);
            quaternion_to_rotvec(aux_g->orientation_g, tmp4);
            // printf("[RAND] %f, %f, %f\n", aux_g->orientation_g[0], aux_g->orientation_g[1], aux_g->orientation_g[2]);
        }

        // generate random uints for all torsions // todo: change sampling of torsions
        for (int i = 0; i < flex_topo.ntorsion; i ++){
            // copy dihedrals
            tmp4[3] = pose_old->dihedrals[i];

            out_pose_new->dihedrals[i] = tmp4[3];
            // set gradient
            ri4.x = hiprand(state);
            ri4.w = ri4.x % flex_topo.range_inds[i * 2 + 1]; // save index of range_list
            ri4.z = flex_topo.range_inds[i * 2] + ri4.w * 2; // save a tmp index
            ri4.x = hiprand(state);

            tmp4[1] = flex_topo.range_list[ri4.z + 1] - flex_topo.range_list[ri4.z];
            ri4.y = ceil(tmp4[1] / TOR_PREC) + 1; // 10 degree as precision

            tmp4[0] = get_real_within_by_int(ri4.x, flex_topo.range_list[ri4.z], flex_topo.range_list[ri4.z + 1], ri4.y);
            aux_g->dihedrals_g[i] = tmp4[0] - tmp4[3];
        }
    }
    tile.sync();

    apply_grad_update_pose_warp(tile, out_pose_new, aux_g, flex_topo, 1.);

}




/**
 * @brief Mutate one pose and update coords.
 * 
 * @param tile Cooperative group
 * @param out_pose Pointer to the pose to be mutated
 * @param flex_topo Topology of the flex
 * @param state cuRand state
 * @param amplitude Amplitude of the mutation
 */
__forceinline__ __device__ void mutate_pose_warp(const cg::thread_block_tile<TILE_SIZE>& tile, FlexPose* out_pose,
                                                 const FlexTopo* flex_topo,
                                                 hiprandStatePhilox4_32_10_t* state, Real amplitude = 1){ //amplitude:2.0
    // DOF, which as an index of DOF
    Real rand_5[5] = {0};
    Real q[4] = {0}; //todo: use rand_5 instead of q to save registers
    Real tmp1[3] = {0};
    Real a = 0;
    int which = -1;

    if (tile.thread_rank() == 0){
        int num_mutable = 2 + flex_topo->ntorsion; //center, orientation, torsions
        if (FLAG_CONSTRAINT_DOCK){
            if (num_mutable < 3){
                which = 3; // no mutation
            } else{
                which = gen_rand_int_within(state, 2, num_mutable - 1);
            }
        }
        else{
            which = gen_rand_int_within(state, 0, num_mutable - 1);
        }
        // DPrint1("which is %d\n", which);
        // prepare random values for choosing DOF to mutate
        gen_4_rand_in_sphere(rand_5, state);
        rand_5[4] = hiprand_uniform(state);
    }
    tile.sync();

    which = tile.shfl(which, 0);
    rand_5[0] = tile.shfl(rand_5[0], 0);
    rand_5[1] = tile.shfl(rand_5[1], 0);
    rand_5[2] = tile.shfl(rand_5[2], 0);
    rand_5[3] = tile.shfl(rand_5[3], 0);
    rand_5[4] = tile.shfl(rand_5[4], 0);

    // 0 for translation
    if (which == 0){
        // compute a translation under box constraint
        tmp1[0] = clamp_by_range(amplitude * rand_5[0] + out_pose->center[0], BOX_X_HI, BOX_X_LO) - out_pose->center[0]; //amplitude * rand_5[0];
        tmp1[1] = clamp_by_range(amplitude * rand_5[1] + out_pose->center[1], BOX_Y_HI, BOX_Y_LO) - out_pose->center[1]; //amplitude * rand_5[1];
        tmp1[2] = clamp_by_range(amplitude * rand_5[2] + out_pose->center[2], BOX_Z_HI, BOX_Z_LO) - out_pose->center[2]; //amplitude * rand_5[2];

        for (int i_at = tile.thread_rank(); i_at < flex_topo->natom; i_at += tile.num_threads()){
            // move to the new center
            out_pose->coords[i_at * 3] = out_pose->coords[i_at * 3] + tmp1[0];
            out_pose->coords[i_at * 3 + 1] = out_pose->coords[i_at * 3 + 1] + tmp1[1];
            out_pose->coords[i_at * 3 + 2] = out_pose->coords[i_at * 3 + 2] + tmp1[2];
        }
        tile.sync();
        if (tile.thread_rank() == 0){
            out_pose->center[0] = out_pose->center[0] + tmp1[0];
            out_pose->center[1] = out_pose->center[1] + tmp1[1];
            out_pose->center[2] = out_pose->center[2] + tmp1[2];
        }
        tile.sync();
    }
    else if (which == 1){
        // 1 for rotation
        if (tile.thread_rank() == 0){
            //1 for rotation of the whole molecule
            a = gyration_radius(out_pose, flex_topo); // an indicator of the size
            if (a > EPSILON){
                // add a random rotation to temporary quaternion
                // the movement step of an atom is roughly amplitude Angstrom
                tmp1[0] = amplitude / a * rand_5[0];
                tmp1[1] = amplitude / a * rand_5[1];
                tmp1[2] = amplitude / a * rand_5[2];

                rotvec_to_quaternion(q, tmp1);
                out_pose->rot_vec[0] = tmp1[0];
                out_pose->rot_vec[1] = tmp1[1];
                out_pose->rot_vec[2] = tmp1[2];
            }
        }
        tile.sync();
        a = tile.shfl(a, 0);

        if (a > EPSILON){
            q[0] = tile.shfl(q[0], 0);
            q[1] = tile.shfl(q[1], 0);
            q[2] = tile.shfl(q[2], 0);
            q[3] = tile.shfl(q[3], 0);

            // rotate all atoms fixme: the rotation has low precision and leads to error over 0.001
            for (int i_at = tile.thread_rank(); i_at < flex_topo->natom; i_at += tile.num_threads()){
                tmp1[0] = out_pose->coords[i_at * 3] - out_pose->center[0];
                tmp1[1] = out_pose->coords[i_at * 3 + 1] - out_pose->center[1];
                tmp1[2] = out_pose->coords[i_at * 3 + 2] - out_pose->center[2];
                rotate_vec_by_quaternion(tmp1, q);
                out_pose->coords[i_at * 3] = tmp1[0] + out_pose->center[0];
                out_pose->coords[i_at * 3 + 1] = tmp1[1] + out_pose->center[1];
                out_pose->coords[i_at * 3 + 2] = tmp1[2] + out_pose->center[2];
            }
            tile.sync();
        }
    }
    else if (which - 2 < flex_topo->ntorsion){
        // rotate one dihedral
        which -= 2;
        // change lig dihedral
        if (tile.thread_rank() == 0){
            a = get_radian_in_ranges(flex_topo->range_list + flex_topo->range_inds[2 * which],
                                     flex_topo->range_inds[2 * which + 1], rand_5 + 3) - out_pose->dihedrals[which];
            // printf("which is %d, a is %f\n", which,  a);
        }
        a = tile.shfl(a, 0); // increment of dihedral value
        apply_grad_update_dihe_warp(tile, out_pose, flex_topo, which, a);
    }
    else{
        // no mutation
        // assert(which - 2 < flex_topo->ntorsion);
    }
}


/**
 * @brief Monte Carlo Kernel. Each warp tackles one pose.
 *
 * @param out_poses Prepared poses that have been initialized, size: nflex * num_pose_per_flex
 * @param flex_topos Topology of flex, size: nflex
 * @param aux_poses Auxiliary poses, size: STRIDE_POSE * nflex * num_pose_per_flex
 * @param aux_gradients Auxiliary gradients, size: STRIDE_G * nflex * num_pose_per_flex
 * @param aux_hessians Auxiliary hessians, size: nflex * num_pose_per_flex
 * @param aux_forces Auxiliary forces, size: nflex * num_pose_per_flex
 * @param states Random states, each pose owns one, size: nflex * num_pose_per_flex
 * @param seed Random seed
 * @param mc_steps Number of MC steps
 * @param opt_steps Number of optimization steps after a pose is accepted
 * @param num_pose_per_flex Number of poses per flex
 * @param max_thread Maximum number of threads used
 */
__global__ void mc_kernel(FlexPose* out_poses, const FlexTopo* flex_topos, const FixMol& fix_mol,
                          const FlexParamVina* flex_params, const FixParamVina& fix_param,
                          FlexPose* aux_poses, FlexPoseGradient* aux_gradients, FlexPoseHessian* aux_hessians,
                          FlexForce* aux_forces,
                          hiprandStatePhilox4_32_10_t* states, int seed, bool randomize,
                          int mc_steps, int opt_steps, int num_pose_per_flex, int max_thread){
    // Just for ONE best pose

    int id_thread = blockIdx.x * blockDim.x + threadIdx.x; // global idx of the thread
    if (id_thread < max_thread){
        int id_pose = id_thread / TILE_SIZE; // 一个pose由1个tile处理，也就是包含多个threads
        int id_flex = id_pose / num_pose_per_flex;

        // Use alias
        FlexPose& out_pose = out_poses[id_pose]; // pointer to global data
        const FlexTopo& flex_topo = flex_topos[id_flex];
        const FlexParamVina& flex_param = flex_params[id_flex];

        FlexPose& pose_candidate = aux_poses[id_pose * STRIDE_POSE];
        FlexPose& pose_accepted = aux_poses[id_pose * STRIDE_POSE + 1];
        FlexPose& aux_pose_ori = aux_poses[id_pose * STRIDE_POSE + 2];
        FlexPose& aux_pose_new = aux_poses[id_pose * STRIDE_POSE + 3];

        FlexPoseGradient& aux_g = aux_gradients[id_pose * STRIDE_G];
        FlexPoseGradient& aux_g_new = aux_gradients[id_pose * STRIDE_G + 1];
        FlexPoseGradient& aux_g_ori = aux_gradients[id_pose * STRIDE_G + 2];
        FlexPoseGradient& aux_p = aux_gradients[id_pose * STRIDE_G + 3];
        FlexPoseGradient& aux_y = aux_gradients[id_pose * STRIDE_G + 4];
        FlexPoseGradient& aux_minus_hy = aux_gradients[id_pose * STRIDE_G + 5];

        FlexPoseHessian& aux_h = aux_hessians[id_pose];

        FlexForce& aux_f = aux_forces[id_pose]; //todo: use struct or just Real* ?

        hiprandStatePhilox4_32_10_t& state = states[id_pose];


        // Cooperative group
        auto cta = cg::this_thread_block();
        cg::thread_block_tile<TILE_SIZE> tile = cg::tiled_partition<TILE_SIZE>(cta);
        // Init hiprand states
        if (tile.thread_rank() == 0){
            hiprand_init(seed, id_pose, 0, &state);
        }
        tile.sync();

        int dim = 3 + 4 + flex_topo.ntorsion;
        Real best_e = 1e9; // large value for finding minimum energy

        if (randomize){
            // prepare the initial pose: each pose is a random pose!
            randomize_pose_warp(tile, &pose_accepted, &aux_g, &out_pose, flex_topo, num_pose_per_flex, &state);
        }else{
            duplicate_pose_warp(tile, &pose_accepted, &out_pose, dim, flex_topo.natom);
        }

        if (mc_steps == 0){
            Real energy = cal_e_f_warp(tile, &pose_accepted, flex_topo, fix_mol, flex_param, fix_param, aux_f.f);

            if (tile.thread_rank() == 0){
                pose_accepted.energy = energy;
            }
            duplicate_pose_warp(tile, &out_pose, &pose_accepted, dim, flex_topo.natom);
        }
        else{
            for (int step = 0; step < mc_steps; step++){
                // 1. mutate conf, PRODUCE a random conf
                DPrint1("========= MC step %d \n", step);

                duplicate_pose_warp(tile, &pose_candidate, &pose_accepted, dim, flex_topo.natom);

                mutate_pose_warp(tile, &pose_candidate, &flex_topo, &state);

                // todo: add clash-detection for efficiency

                // Record initial energy and gradient. E_ori is energy, aux_g is set as current gradient
                // If max_steps == 0, this func only records the energy of original structure.
                if (opt_steps == 0){
                    Real energy = cal_e_grad_warp(tile, &pose_candidate, &aux_g, flex_topo, fix_mol,
                                                  flex_param, fix_param, aux_f.f);

                    if (tile.thread_rank() == 0){
                        pose_candidate.energy = energy;
                    }
                    tile.sync();
                }
                else{
                    // essential optimization. only computes the energy of candidate pose for MC task
                    // coords are updated inside bfgs
                    bfgs_warp(tile,
                              &pose_candidate, flex_topo, fix_mol,
                              flex_param, fix_param,
                              &aux_pose_new, &aux_pose_ori,
                              &aux_g, &aux_g_new, &aux_g_ori,
                              &aux_p, &aux_y, &aux_minus_hy,
                              &aux_h, &aux_f, opt_steps);
                }


                // 2. Metropolis
                bool accepted = false;
                if (tile.thread_rank() == 0){
                    Real rand_x = hiprand_uniform(&states[id_pose]);
                    accepted = metropolis_accept(pose_accepted.energy, pose_candidate.energy, 1.2, rand_x);
                }
                accepted = tile.shfl(accepted, 0);

                // if accepted
                if (step == 0 || accepted){
                    // set accepted pose as this lately accepted candidate
                    duplicate_pose_warp(tile, &pose_accepted, &pose_candidate, dim, flex_topo.natom);

                    // Possibly the best pose by now
                    if (pose_accepted.energy < best_e){
                        duplicate_pose_warp(tile, &out_pose, &pose_accepted, dim, flex_topo.natom);
                        best_e = pose_accepted.energy;
                    }
                }
            }
        }
    }
}


void mc_cu(FlexPose* out_poses, const FlexTopo* topos,
           const FixMol& fix_mol, const FlexParamVina* flex_param, const FixParamVina& fix_param,
           FlexPose* aux_poses, FlexPoseGradient* aux_gradients, FlexPoseHessian* aux_hessians, FlexForce* aux_forces,
           int mc_steps, int opt_steps, int nflex, int exhuastiveness, int seed, bool randomize){
    //------- perform MC on GPU -------//

    const int block_size = TILE_SIZE; // One block for one tile (for 32, namely one warp per block)
    int npose = nflex * exhuastiveness;

    // initilize hiprand states
    hiprandStatePhilox4_32_10_t* states;
    checkCUDA(hipMalloc(&states, sizeof(hiprandStatePhilox4_32_10_t) * npose));

    // run the kernel
    mc_kernel<<<npose, block_size>>>(out_poses, topos, fix_mol,
                                     flex_param, fix_param,
                                     aux_poses, aux_gradients, aux_hessians, aux_forces,
                                     states, seed, randomize,
                                     mc_steps, opt_steps, exhuastiveness, npose * block_size);
    checkCUDA(hipDeviceSynchronize());
    spdlog::warn("[Line Search Steps Count]: {}", funcCallCount);

    // free mem
    checkCUDA(hipFree(states));
}
