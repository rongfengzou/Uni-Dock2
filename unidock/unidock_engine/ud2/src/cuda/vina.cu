//
// Created by Congcong Liu on 24-12-9.
//


#include "score/vina.h"

// ==================  Memory allocation for ONE object ==================
FlexParamVina* alloccp_FlexParamVina_gpu(const FlexParamVina& flex_param_vina, int natom){
    FlexParamVina* flex_param_vina_cu;
    hipMalloc(&flex_param_vina_cu, sizeof(FlexParamVina));

    FlexParamVina flex_param_vina_tmp;
    flex_param_vina_tmp.npair_intra = flex_param_vina.npair_intra;
    flex_param_vina_tmp.npair_inter = flex_param_vina.npair_inter;
    hipMalloc(&flex_param_vina_tmp.pairs_intra, sizeof(int) * flex_param_vina.npair_intra * 2);
    hipMemcpy(flex_param_vina_tmp.pairs_intra, flex_param_vina.pairs_intra, sizeof(int) * flex_param_vina.npair_intra * 2, hipMemcpyHostToDevice);
    hipMalloc(&flex_param_vina_tmp.r1_plus_r2_intra, sizeof(Real) * flex_param_vina.npair_intra);
    hipMemcpy(flex_param_vina_tmp.r1_plus_r2_intra, flex_param_vina.r1_plus_r2_intra, sizeof(Real) * flex_param_vina.npair_intra, hipMemcpyHostToDevice);
    hipMalloc(&flex_param_vina_tmp.pairs_inter, sizeof(int) * flex_param_vina.npair_inter * 2);
    hipMemcpy(flex_param_vina_tmp.pairs_inter, flex_param_vina.pairs_inter, sizeof(int) * flex_param_vina.npair_inter * 2, hipMemcpyHostToDevice);
    hipMalloc(&flex_param_vina_tmp.r1_plus_r2_inter, sizeof(Real) * flex_param_vina.npair_inter);
    hipMemcpy(flex_param_vina_tmp.r1_plus_r2_inter, flex_param_vina.r1_plus_r2_inter, sizeof(Real) * flex_param_vina.npair_inter, hipMemcpyHostToDevice);
    hipMalloc(&flex_param_vina_tmp.atom_types, sizeof(int) * natom);
    hipMemcpy(flex_param_vina_tmp.atom_types, flex_param_vina.atom_types, sizeof(int) * natom, hipMemcpyHostToDevice);

    hipMemcpy(flex_param_vina_cu, &flex_param_vina_tmp, sizeof(FlexParamVina), hipMemcpyHostToDevice);
    return flex_param_vina_cu;
}
void free_FlexParamVina_gpu(FlexParamVina* flex_param_vina_cu){
    FlexParamVina flex_param_vina_tmp;
    hipMemcpy(&flex_param_vina_tmp, flex_param_vina_cu, sizeof(FlexParamVina), hipMemcpyDeviceToHost);
    hipFree(flex_param_vina_tmp.atom_types);
    hipFree(flex_param_vina_tmp.pairs_intra);
    hipFree(flex_param_vina_tmp.r1_plus_r2_intra);
    hipFree(flex_param_vina_tmp.pairs_inter);
    hipFree(flex_param_vina_tmp.r1_plus_r2_inter);
    hipFree(flex_param_vina_cu);
}


FixParamVina* alloccp_FixParamVina_gpu(const FixParamVina& fix_param_vina, int natom){
    FixParamVina* fix_param_vina_cu;
    hipMalloc(&fix_param_vina_cu, sizeof(FixParamVina));

    FixParamVina fix_param_vina_tmp;
    hipMalloc(&fix_param_vina_tmp.atom_types, sizeof(int) * natom);
    hipMemcpy(fix_param_vina_tmp.atom_types, fix_param_vina.atom_types, sizeof(int) * natom, hipMemcpyHostToDevice);

    hipMemcpy(fix_param_vina_cu, &fix_param_vina_tmp, sizeof(FixParamVina), hipMemcpyHostToDevice);
    return fix_param_vina_cu;
}
void free_FixParamVina_gpu(FixParamVina* fix_param_vina_cu){
    FixParamVina fix_param_vina_tmp;
    hipMemcpy(&fix_param_vina_tmp, fix_param_vina_cu, sizeof(FixParamVina), hipMemcpyDeviceToHost);
    hipFree(fix_param_vina_tmp.atom_types);
    hipFree(fix_param_vina_cu);
}
